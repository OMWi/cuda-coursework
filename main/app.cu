#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <algorithm>
#include "hip/hip_runtime.h"
#include ""

using namespace std;

#define idx(c, x, y, C, X, Y) ((x) + (X) * ((y) + (Y) * (c)))
// #define f_idx(a, c, x, y, A, C, X, Y) ((x) + (X) * ((y) + (Y) * ((c) + (C) * (a))))
#define f_idx(inpCh, outCh, InpCh) ((inpCh) + (InpCh*outCh))
#define ReLU(v) (max((v), 0.0f))

#define idx2(ch, row, col, size) ((ch)*(size)*(size) + (row)*(size) + col)


__shared__ float buffer[];
__global__ void depthwise_separable_convolution(
      int inputSize, int inputChannels,
      int outputChannels, const float* input, float* output, 
      const float* filter1, const float* filter2, int filterSize
) {
   const int n2y = blockIdx.x;
   const int n2x = blockIdx.y;
   const int ch = threadIdx.x;

   if (ch > inputChannels) return;
   float t = 0;
   for (int f1y = 0; f1y < filterSize; f1y++){
      for (int f1x = 0; f1x < filterSize; f1x++) {
         t += input[idx(ch, n2x + f1x, n2y + f1y, inputChannels, inputSize, inputSize)] * filter1[idx(ch, f1x, f1y, inputChannels, filterSize, filterSize)];
      }
   }
   buffer[ch] = t;

   __syncthreads();

   if (ch >= outputChannels) return;
   t = 0;
   for (int ch2 = 0; ch2 < inputChannels; ch2++) {
      t += buffer[ch2] * filter2[f_idx(ch, ch2, inputChannels)];
   }
   output[idx(ch, n2x, n2y, outputChannels, inputSize, inputSize)] = ReLU(t);
}

__global__ void depthwise_separable_convolution_toeplitz(
      int inputSize, int inputChannels,
      int outputChannels, const float* input, float* output, 
      const float* filter1, const float* filter2, int filterSize
) {
   const int n2y = blockIdx.x;
   const int n2x = blockIdx.y;
   const int ch = threadIdx.x;

   if (ch > inputChannels) return;
   int toeplitzRowIdx = n2y*inputSize + n2x;
   int groupSizeX = inputSize+1;
   int groupSizeY = inputSize+1 - filterSize + 1;
   // int groupIdxY = toeplitzRowIdx / groupSizeY;
   // int groupIdxX = groupIdxY;
   int groupIdx = toeplitzRowIdx / groupSizeY;
   int zeroesLeft = toeplitzRowIdx % groupSizeY;
   // int zeroesRight = groupSizeX - filterSize - zeroesLeft;  

   float t = 0;
   for (int curGroup = groupIdx, fRow = 0; fRow < filterSize; curGroup++, fRow++) {
      
      for (int j = 0; j < filterSize; j++) {
         if (groupIdx == 0 && ch == 0) {
         printf("curGroup %d;  fRow %d;  j %d;\t f=%d;  i=%d\n", curGroup, fRow, j, filter1[idx2(ch, fRow, j, filterSize)], input[idx2(ch, curGroup, j, inputSize+1)]);
      }
         t += filter1[idx2(ch, fRow, j, filterSize)] * input[idx2(ch, curGroup,  j, inputSize+1)]; // ?
      }
   }
   // for (int i = groupIdx*groupSizeX + zeroesLeft, curGroup = groupIdx; i < inputSize; i += groupSizeX, curGroup++) {
   //    for (int j = 0; j < filterSize; j++) {
   //       int inpIdx = ch*inputSize*inputSize + curGroup*inputSize + j;
   //       t += filter1[idx2(ch, curGroup, j, filterSize)] * input[toeplitzRowIdx]; // ?
   //    }
   // }
   if (ch == 0 && groupIdx == 0) 
      printf("ch=%d; n2y=%d; n2x=%d; grIdx=%d; tRow=%d;grSize=(%d, %d)\t%d\n\n", ch, n2y, n2x, groupIdx, 
         toeplitzRowIdx, groupSizeX, groupSizeY, t);
   buffer[ch] = t;

   __syncthreads();

   if (ch >= outputChannels) return;
   t = 0;
   for (int ch2 = 0; ch2 < inputChannels; ch2++) {
      t += buffer[ch2] * filter2[f_idx(ch, ch2, inputChannels)];
   }
   output[idx(ch, n2x, n2y, outputChannels, inputSize, inputSize)] = ReLU(t);
}

bool check_error_status(hipError_t status, const char *error_message) {
    if (status != hipSuccess) {
        fprintf(stderr, error_message);
        return true;
    }
    return false;
}

bool test(int inputSize, int inputChannels, int outputChannels, int filterSize) {
   hipError_t status;  
   // printf("test start\n");

   status = hipSetDevice(0);
   if (check_error_status(status, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?\n"))
      return false;

      
   vector<float> hInput(inputChannels*inputSize*inputSize);
   vector<float> hOutput(outputChannels*inputSize*inputSize);
   vector<float> hFilter1(inputChannels*filterSize*filterSize);
   vector<float> hFilter2(inputChannels*outputChannels);

   for (int ch = 0; ch < inputChannels; ch++) {
      for (int y = 0; y < inputSize; y++) {
         for (int x = 0; x < inputSize; x++) {
            hInput[idx(ch, x, y, inputChannels, inputSize, inputSize)] = rand() % 3 + 1.0 / (1.0 + rand() % 3);
         }
      }
   }
   // printf("hInput +\n");
   for (int ch = 0; ch < inputChannels; ch++) {
      for (int y = 0; y < filterSize; y++) {
         for (int x = 0; x < filterSize; x++) {
            hFilter1[idx(ch, x, y, inputChannels, filterSize, filterSize)] = rand() % 3 + 1.0 / (1.0 + rand() % 3);
         }
      }
   }
   // printf("hFilter1 +\n");
   for (int oChannel = 0; oChannel < outputChannels; oChannel++) {
      for (int iChannel = 0; iChannel < inputChannels; iChannel++) {
         hFilter2[f_idx(iChannel, oChannel, inputChannels)] = rand() % 3 + 1.0 / (1.0 + rand() % 3);
      }
   }
   // printf("host data initialized\n");
   

   float *dInput, *dOutput, *dFilter1, *dFilter2;

   status = hipMalloc((void**)&dInput, hInput.size()*sizeof(float));
   if (check_error_status(status, "hipMalloc failed!\n"))
      return false;
   status = hipMalloc((void**)&dOutput, hOutput.size()*sizeof(float));
   if (check_error_status(status, "hipMalloc failed!\n"))
      return false;
   status = hipMalloc((void**)&dFilter1, hFilter1.size()*sizeof(float));
   if (check_error_status(status, "hipMalloc failed!\n"))
      return false;
   status = hipMalloc((void**)&dFilter2, hFilter2.size()*sizeof(float));
   if (check_error_status(status, "hipMalloc failed!\n"))
      return false;

   // printf("device data malloc success\n");

   status = hipMemcpy(dInput, hInput.data(), hInput.size()*sizeof(float), hipMemcpyHostToDevice);
   if (check_error_status(status, "cudaMemCpy failed!\n"))
      return false;
   status = hipMemcpy(dFilter1, hFilter1.data(), hFilter1.size()*sizeof(float), hipMemcpyHostToDevice);
   if (check_error_status(status, "cudaMemCpy failed!\n"))
      return false;
   status = hipMemcpy(dFilter2, hFilter2.data(), hFilter2.size()*sizeof(float), hipMemcpyHostToDevice);
   if (check_error_status(status, "cudaMemCpy failed!\n"))
      return false;

   // printf("device data memcpy success\n");

   // int blockSizeX = inputSize;
   // int blockSizeY = inputSize - filterSize + 1;
   // vector<float> hToeplitz(inputChannels*blockSizeX*blockSizeX*blockSizeY*blockSizeY);

   dim3 dimBlock(max(inputChannels, outputChannels), 1);
   dim3 dimGrid(inputSize, inputSize);
   // printf("foo started\n");
   depthwise_separable_convolution<<<dimGrid, dimBlock, dimBlock.x * sizeof(float)>>>(
      inputSize, inputChannels,
      outputChannels, dInput, dOutput, 
      dFilter1, dFilter2, filterSize);
   // printf("foo ended\n");

   status = hipMemcpy(hOutput.data(), dOutput, hOutput.size()*sizeof(float), hipMemcpyDeviceToHost);
   if (check_error_status(status, "couldn't load device output to host"))
      return false;
   // printf("output copy success\n");

   hipFree(dInput);
   hipFree(dOutput);
   hipFree(dFilter1);
   hipFree(dFilter2);

   return true;
}


void print_vector2d(vector<float> vec, int size, int channel) {
   printf("Channel %d\n", channel);
   for (int y = 0; y < size; y++) {
      for (int x = 0; x < size; x++) {
         printf("%f ", vec[idx2(channel, y, x, size)]);
      }
      printf("\n");
   }
}

bool static_test() {
   int inputChannels = 2, outputChannels = 2;
   int inputSize = 3, filterSize = 2;
   hipError_t status;  

   status = hipSetDevice(0);
   if (check_error_status(status, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?\n"))
      return false;
      
   vector<float> hInput(inputChannels*(inputSize+1)*(inputSize+1));
   vector<float> hOutput(outputChannels*inputSize*inputSize);
   vector<float> hFilter1(inputChannels*filterSize*filterSize);
   vector<float> hFilter2(inputChannels*outputChannels);

   for (int ch = 0; ch < inputChannels; ch++) {
      for (int y = 0; y < inputSize; y++) {
         for (int x = 0; x < inputSize; x++) {
            hInput[idx2(ch, y, x, inputSize+1)] = rand() % 3 + 1.0 / (1.0 + rand() % 3);
         }
      }
   }
   printf("hInput+\n");
   for (int ch = 0; ch < inputChannels; ch++) {
      for (int x = 0, y = inputSize; x < inputSize+1; x++) {
         hInput[idx2(ch, y, x, inputSize+1)] = 0;         
      }
      for (int x = inputSize, y = 0; y < inputSize+1; y++) {
         hInput[idx2(ch, y, x, inputSize+1)] = 0;         
      }
   }
   printf("padding+\n");
   printf("Input:\n");
   for (int ch = 0; ch < inputChannels; ch++) {
      print_vector2d(hInput, inputSize+1, ch);
   }
   // printf("hInput +\n");
   for (int ch = 0; ch < inputChannels; ch++) {
      for (int y = 0; y < filterSize; y++) {
         for (int x = 0; x < filterSize; x++) {
            hFilter1[idx(ch, x, y, inputChannels, filterSize, filterSize)] = rand() % 3 + 1.0 / (1.0 + rand() % 3);
         }
      }
   }
   printf("Filter 1:\n");
   for (int ch = 0; ch < inputChannels; ch++) {
      print_vector2d(hFilter1, filterSize, ch);
   }
   // printf("hFilter1 +\n");
   for (int oChannel = 0; oChannel < outputChannels; oChannel++) {
      for (int iChannel = 0; iChannel < inputChannels; iChannel++) {
         hFilter2[f_idx(iChannel, oChannel, inputChannels)] = rand() % 3 + 1.0 / (1.0 + rand() % 3);
      }
   }
   // printf("host data initialized\n");
   

   float *dInput, *dOutput, *dFilter1, *dFilter2;

   status = hipMalloc((void**)&dInput, hInput.size()*sizeof(float));
   if (check_error_status(status, "hipMalloc failed!\n"))
      return false;
   status = hipMalloc((void**)&dOutput, hOutput.size()*sizeof(float));
   if (check_error_status(status, "hipMalloc failed!\n"))
      return false;
   status = hipMalloc((void**)&dFilter1, hFilter1.size()*sizeof(float));
   if (check_error_status(status, "hipMalloc failed!\n"))
      return false;
   status = hipMalloc((void**)&dFilter2, hFilter2.size()*sizeof(float));
   if (check_error_status(status, "hipMalloc failed!\n"))
      return false;

   // printf("device data malloc success\n");

   status = hipMemcpy(dInput, hInput.data(), hInput.size()*sizeof(float), hipMemcpyHostToDevice);
   if (check_error_status(status, "cudaMemCpy failed!\n"))
      return false;
   status = hipMemcpy(dFilter1, hFilter1.data(), hFilter1.size()*sizeof(float), hipMemcpyHostToDevice);
   if (check_error_status(status, "cudaMemCpy failed!\n"))
      return false;
   status = hipMemcpy(dFilter2, hFilter2.data(), hFilter2.size()*sizeof(float), hipMemcpyHostToDevice);
   if (check_error_status(status, "cudaMemCpy failed!\n"))
      return false;

   // printf("device data memcpy success\n");

   // int blockSizeX = inputSize;
   // int blockSizeY = inputSize - filterSize + 1;
   // vector<float> hToeplitz(inputChannels*blockSizeX*blockSizeX*blockSizeY*blockSizeY);

   dim3 dimBlock(max(inputChannels, outputChannels), 1);
   dim3 dimGrid(inputSize, inputSize);
   // printf("foo started\n");
   depthwise_separable_convolution_toeplitz<<<dimGrid, dimBlock, dimBlock.x * sizeof(float)>>>(
      inputSize, inputChannels,
      outputChannels, dInput, dOutput, 
      dFilter1, dFilter2, filterSize);
   // printf("foo ended\n");

   status = hipMemcpy(hOutput.data(), dOutput, hOutput.size()*sizeof(float), hipMemcpyDeviceToHost);
   if (check_error_status(status, "couldn't load device output to host"))
      return false;

   printf("Output:\n");
   for (int ch = 0; ch < outputChannels; ch++) {
      print_vector2d(hOutput, inputSize, ch);
   }
   // printf("output copy success\n");

   hipFree(dInput);
   hipFree(dOutput);
   hipFree(dFilter1);
   hipFree(dFilter2);

   return true;
}

int main() {
   int inputSize = 1 << 10;
	int inputChannels = 3;
	int outputChannels = 16;
	int filterSize = 3;

   static_test();

   // int testNum = 1000;   
   // for (int i = 0; i < testNum; i++) {
   //    bool res = test(inputSize, inputChannels, outputChannels, filterSize);
   //    if (res == false) {
   //       printf("%d test fail\n", i);
   //       return -1;
   //    }
   //    printf("%d test success\n", i);
   // }   
}