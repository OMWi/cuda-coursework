#include "hip/hip_runtime.h"
#include <stdio.h>
#include <vector>
#include <iostream>
#include <fstream>
#include <algorithm>
// #include "hip/hip_runtime.h"
// #include ""

using namespace std;

// #define f_idx(a, c, x, y, A, C, X, Y) ((x) + (X) * ((y) + (Y) * ((c) + (C) * (a))))

#define ReLU(v) (max((v), 0.0f))

#define idx(ch, row, col, size) ((ch)*(size)*(size) + (row)*(size) + col)
#define f_idx(inpCh, outCh, InpCh) ((inpCh) + (InpCh*outCh))

void printVector2d(vector<float> vec, int size, int channel) {
   printf("Channel %d\n", channel);
   for (int y = 0; y < size; y++) {
      for (int x = 0; x < size; x++)
         printf("%f ", vec[idx(channel, y, x, size)]);
      printf("\n");
   }
   printf("\n");
}

void printFilter2(vector<float> f, int inputChannels, int outputChannels) {
   printf("Filter 2:\n");
   for (int o = 0; o < outputChannels; o++) {
      for (int i = 0; i < inputChannels; i++)
         printf("%f ", f[f_idx(i, o, inputChannels)]);
      printf("\n");
   }
   printf("\n");
}

__shared__ float buffer[];
__global__ void depthwise_separable_convolution(
      int inputSize, int inputChannels,
      int outputChannels, const float* input, float* output, 
      const float* filter1, const float* filter2, int filterSize
) {
   const int x = blockIdx.x;
   const int y = blockIdx.y;
   const int ch = threadIdx.x;
   float t;

   if (ch < inputChannels) {
      t = 0;
      for (int f1y = 0; f1y < filterSize; f1y++){
         for (int f1x = 0; f1x < filterSize; f1x++) {
            t += input[idx(ch, y + f1y, x + f1x, inputSize+2)] * filter1[idx(ch, f1y, f1x, filterSize)];
         }
      }
      buffer[ch] = t;
   }   

   __syncthreads();

   if (ch < outputChannels) {
      t = 0;
      for (int inpCh = 0; inpCh < inputChannels; inpCh++) {
         t += buffer[ch] * filter2[f_idx(inpCh, ch, inputChannels)];
      }
      output[idx(ch, y, x, inputSize)] = ReLU(t);
   }   
}


__shared__ float tBuffer[];
__global__ void depthwise_separable_convolution_toeplitz(
      int inputSize, int inputChannels,
      int outputChannels, const float* input, float* output, 
      const float* filter1, const float* filter2, int filterSize
) {
   const int x = blockIdx.x;
   const int y = blockIdx.y;
   const int ch = threadIdx.x;
   float t;

   if (ch < inputChannels) {
      // int toeplitzRowIdx = 
      // int groupSizeX = inputSize+2;
      int groupSizeY = inputSize+2 - filterSize + 1;
      int startGroupIdx = (y*inputSize + x) / groupSizeY;
      int zeroesLeft = (y*inputSize + x) % groupSizeY;

      t = 0;
      for (int fRow = 0; fRow < filterSize; fRow++) {
         int inpRow = startGroupIdx + fRow;
         for (int fCol = 0; fCol < filterSize; fCol++) {            
            // int inpCol = ;
            t += filter1[idx(ch, fRow, fCol, filterSize)] * input[idx(ch, inpRow, zeroesLeft + fCol, inputSize+2)];
         }
      }
      tBuffer[ch] = t;
   }   

   __syncthreads();

   if (ch < outputChannels) {
      t = 0;
      for (int inpCh = 0; inpCh < inputChannels; inpCh++) {
         t += tBuffer[ch] * filter2[f_idx(inpCh, ch, inputChannels)];
      }
      output[idx(ch, y, x, inputSize)] = ReLU(t);
   }   
}


bool check_error_status(hipError_t status, const char *error_message) {
   if (status != hipSuccess) {
      fprintf(stderr, error_message);
      return true;
   }
   return false;
}

bool test(int inputSize, int inputChannels, int outputChannels, int filterSize,
      vector<float> &hInput, vector<float> &hOutput, vector<float> &hFilter1, vector<float> &hFilter2
      // , float &funcTime, float &memTime
) {   
   hipError_t status; 
   // hipEvent_t funcStart, funcStop, memStart, memStop;
   // hipEventCreate(&funcStart); 
   // hipEventCreate(&funcStop);
   // hipEventCreate(&memStart);
   // hipEventCreate(&memStop);

   status = hipSetDevice(0);
   if (check_error_status(status, "hipSetDevice fail\n")) return false;

   // hipEventRecord(memStart);
   float *dInput, *dOutput, *dFilter1, *dFilter2;
   status = hipMalloc((void**)&dInput, hInput.size()*sizeof(float));
   if (check_error_status(status, "hipMalloc fail\n"))
      return false;
   status = hipMalloc((void**)&dOutput, hOutput.size()*sizeof(float));
   if (check_error_status(status, "hipMalloc fail\n"))
      return false;
   status = hipMalloc((void**)&dFilter1, hFilter1.size()*sizeof(float));
   if (check_error_status(status, "hipMalloc fail\n"))
      return false;
   status = hipMalloc((void**)&dFilter2, hFilter2.size()*sizeof(float));
   if (check_error_status(status, "hipMalloc fail\n"))
      return false;

   status = hipMemcpy(dInput, hInput.data(), hInput.size()*sizeof(float), hipMemcpyHostToDevice);
   if (check_error_status(status, "cudaMemCpy fail\n"))
      return false;
   status = hipMemcpy(dFilter1, hFilter1.data(), hFilter1.size()*sizeof(float), hipMemcpyHostToDevice);
   if (check_error_status(status, "cudaMemCpy fail\n"))
      return false;
   status = hipMemcpy(dFilter2, hFilter2.data(), hFilter2.size()*sizeof(float), hipMemcpyHostToDevice);
   if (check_error_status(status, "cudaMemCpy fail\n"))
      return false;

   // hipEventRecord(memStop);
   // hipEventSynchronize(memStop);
   // float temp;
   // hipEventElapsedTime(&temp, memStart, memStop);
   // memTime += temp;

   dim3 dimBlock(max(inputChannels, outputChannels), 1);
   dim3 dimGrid(inputSize, inputSize);
   depthwise_separable_convolution<<<dimGrid, dimBlock, dimBlock.x * sizeof(float)>>>(
      inputSize, inputChannels,
      outputChannels, dInput, dOutput, 
      dFilter1, dFilter2, filterSize
   );

   // hipEventRecord(funcStop);
   // hipEventSynchronize(funcStop);
   // hipEventElapsedTime(&temp, funcStart, funcStop);
   // funcTime += temp;

   // hipEventRecord(memStart);
   status = hipMemcpy(hOutput.data(), dOutput, hOutput.size()*sizeof(float), hipMemcpyDeviceToHost);
   if (check_error_status(status, "couldn't load device output to host"))
      return false;
   // hipEventRecord(memStop);
   // hipEventSynchronize(memStop);
   // hipEventElapsedTime(&temp, memStart, memStop);
   // memTime += temp;

   hipFree(dInput);
   hipFree(dOutput);
   hipFree(dFilter1);
   hipFree(dFilter2);
   // hipEventDestroy(funcStart);
   // hipEventDestroy(funcStop);
   // hipEventDestroy(memStart);
   // hipEventDestroy(memStop);
   return true;
}

bool test_toeplitz(int inputSize, int inputChannels, int outputChannels, int filterSize,
      vector<float> &hInput, vector<float> &hOutput, vector<float> &hFilter1, vector<float> &hFilter2
      // , float &funcTime, float &memTime
) {
   hipError_t status;
   // hipEvent_t funcStart, funcStop, memStart, memStop;
   // hipEventCreate(&funcStart); 
   // hipEventCreate(&funcStop);
   // hipEventCreate(&memStart);
   // hipEventCreate(&memStop);

   

   status = hipSetDevice(0);
   if (check_error_status(status, "hipSetDevice fail\n")) return false;

   // hipEventRecord(memStart);
   float *dInput, *dOutput, *dFilter1, *dFilter2;
   status = hipMalloc((void**)&dInput, hInput.size()*sizeof(float));
   if (check_error_status(status, "hipMalloc fail\n"))
      return false;
   status = hipMalloc((void**)&dOutput, hOutput.size()*sizeof(float));
   if (check_error_status(status, "hipMalloc fail\n"))
      return false;
   status = hipMalloc((void**)&dFilter1, hFilter1.size()*sizeof(float));
   if (check_error_status(status, "hipMalloc fail\n"))
      return false;
   status = hipMalloc((void**)&dFilter2, hFilter2.size()*sizeof(float));
   if (check_error_status(status, "hipMalloc fail\n"))
      return false;

   status = hipMemcpy(dInput, hInput.data(), hInput.size()*sizeof(float), hipMemcpyHostToDevice);
   if (check_error_status(status, "cudaMemCpy fail\n"))
      return false;
   status = hipMemcpy(dFilter1, hFilter1.data(), hFilter1.size()*sizeof(float), hipMemcpyHostToDevice);
   if (check_error_status(status, "cudaMemCpy fail\n"))
      return false;
   status = hipMemcpy(dFilter2, hFilter2.data(), hFilter2.size()*sizeof(float), hipMemcpyHostToDevice);
   if (check_error_status(status, "cudaMemCpy fail\n"))
      return false;
   
   // hipEventRecord(memStop);
   // hipEventSynchronize(memStop);
   // float temp;
   // hipEventElapsedTime(&temp, memStart, memStop);
   // memTime += temp;

   // hipEventRecord(funcStart);
   dim3 dimBlock(max(inputChannels, outputChannels), 1);
   dim3 dimGrid(inputSize, inputSize);
   depthwise_separable_convolution_toeplitz<<<dimGrid, dimBlock, dimBlock.x * sizeof(float)>>>(
      inputSize, inputChannels,
      outputChannels, dInput, dOutput, 
      dFilter1, dFilter2, filterSize
   );

   // hipEventRecord(funcStop);
   // hipEventSynchronize(funcStop);
   // hipEventElapsedTime(&temp, funcStart, funcStop);
   // funcTime += temp;

   // hipEventRecord(memStart);
   status = hipMemcpy(hOutput.data(), dOutput, hOutput.size()*sizeof(float), hipMemcpyDeviceToHost);
   if (check_error_status(status, "couldn't load device output to host"))
      return false;
   // hipEventRecord(memStop);
   // hipEventSynchronize(memStop);
   // hipEventElapsedTime(&temp, memStart, memStop);
   // memTime += temp;

   hipFree(dInput);
   hipFree(dOutput);
   hipFree(dFilter1);
   hipFree(dFilter2);
   // hipEventDestroy(funcStart);
   // hipEventDestroy(funcStop);
   // hipEventDestroy(memStart);
   // hipEventDestroy(memStop);
   return true;
}


bool fillInput(vector<float> &inp, int channelNum, int inpSize) {
   if (inp.size() < channelNum*(inpSize+2)*(inpSize+2)){
      printf("Couldn't fill input vector\n");
      return false;
   }
   for (int ch = 0; ch < channelNum; ch++) {
      for (int y = 0; y < inpSize+2; y++)
         for (int x = 0; x < inpSize+2; x++)
            inp[idx(ch, y, x, inpSize+2)] = rand() % 3 + 1.0 / (1.0 + rand() % 3);
      for (int x = 0, y = 0; x < inpSize+2; x++)
         inp[idx(ch, y, x, inpSize+2)] = 0;
      for (int x = 0, y = inpSize+1; x < inpSize+2; x++)
         inp[idx(ch, y, x, inpSize+2)] = 0;   

      for (int x = 0, y = 0; y < inpSize+2; y++)
         inp[idx(ch, y, x, inpSize+2)] = 0;
      for (int x = inpSize+1, y = 0; y < inpSize+2; y++)
         inp[idx(ch, y, x, inpSize+2)] = 0;   
   }  
   return true;
}

bool fillFilter1(vector<float> &f, int channelNum, int dimSize) {
   if (f.size() < channelNum*dimSize*dimSize) {
      printf("Couldn't fill filter 1\n");
      return false;
   }
   for (int ch = 0; ch < channelNum; ch++) {
      for (int y = 0; y < dimSize; y++)
         for (int x = 0; x < dimSize; x++)
            f[idx(ch, y, x, dimSize)] = rand() % 3 + 1.0 / (1.0 + rand() % 3);
   }
   return true;   
}

bool fillFilter2(vector<float> &f, int inputChannels, int outputChannels) {
   if (f.size() < inputChannels*outputChannels) {
      printf("Couldn't fill filter 2\n");
      return false;
   }
   for (int oChannel = 0; oChannel < outputChannels; oChannel++)
      for (int iChannel = 0; iChannel < inputChannels; iChannel++)
         f[f_idx(iChannel, oChannel, inputChannels)] = rand() % 3 + 1.0 / (1.0 + rand() % 3);
   return true;
}


bool mini_test() {
   const int inputSize = 4;
	const int inputChannels = 2;
	const int outputChannels = 2;
	const int filterSize = 3;
   hipError_t status; 
   printf("Mini test:\n"); 

   status = hipSetDevice(0);
   if (check_error_status(status, "hipSetDevice fail\n")) return false;
      
   vector<float> hInput(inputChannels*(inputSize+2)*(inputSize+2));
   vector<float> hOutput(outputChannels*inputSize*inputSize);
   vector<float> hFilter1(inputChannels*filterSize*filterSize);
   vector<float> hFilter2(inputChannels*outputChannels);
   fillInput(hInput, inputChannels, inputSize);
   printf("Input:\n");
   printVector2d(hInput, inputSize+2, 0);
   // printVector2d(hInput, inputSize+2, 1);

   fillFilter1(hFilter1, inputChannels, filterSize);
   printf("Filter 1:\n");
   printVector2d(hFilter1, filterSize, 0);
   // printVector2d(hFilter1, filterSize, 1);

   fillFilter2(hFilter2, inputChannels, outputChannels);
   printFilter2(hFilter2, inputChannels, outputChannels);

   float *dInput, *dOutput, *dFilter1, *dFilter2;
   status = hipMalloc((void**)&dInput, hInput.size()*sizeof(float));
   if (check_error_status(status, "hipMalloc fail\n"))
      return false;
   status = hipMalloc((void**)&dOutput, hOutput.size()*sizeof(float));
   if (check_error_status(status, "hipMalloc fail\n"))
      return false;
   status = hipMalloc((void**)&dFilter1, hFilter1.size()*sizeof(float));
   if (check_error_status(status, "hipMalloc fail\n"))
      return false;
   status = hipMalloc((void**)&dFilter2, hFilter2.size()*sizeof(float));
   if (check_error_status(status, "hipMalloc fail\n"))
      return false;
   // printf("cuda malloc done\n");

   status = hipMemcpy(dInput, hInput.data(), hInput.size()*sizeof(float), hipMemcpyHostToDevice);
   if (check_error_status(status, "cudaMemCpy fail\n"))
      return false;
   status = hipMemcpy(dFilter1, hFilter1.data(), hFilter1.size()*sizeof(float), hipMemcpyHostToDevice);
   if (check_error_status(status, "cudaMemCpy fail\n"))
      return false;
   status = hipMemcpy(dFilter2, hFilter2.data(), hFilter2.size()*sizeof(float), hipMemcpyHostToDevice);
   if (check_error_status(status, "cudaMemCpy fail\n"))
      return false;
   // printf("cuda memcpy done\n");

   dim3 dimBlock(max(inputChannels, outputChannels), 1);
   dim3 dimGrid(inputSize, inputSize);
   // printf("func started\n");
   depthwise_separable_convolution<<<dimGrid, dimBlock, dimBlock.x * sizeof(float)>>>(
      inputSize, inputChannels,
      outputChannels, dInput, dOutput, 
      dFilter1, dFilter2, filterSize
   );
   // printf("func ended\n");

   status = hipMemcpy(hOutput.data(), dOutput, hOutput.size()*sizeof(float), hipMemcpyDeviceToHost);
   if (check_error_status(status, "couldn't load device output to host"))
      return false;
   printf("Output standard:\n");
   printVector2d(hOutput, inputSize, 0);
   // printVector2d(hOutput, inputSize, 1);

   // printf("toeplitz started\n");
   depthwise_separable_convolution_toeplitz<<<dimGrid, dimBlock, dimBlock.x * sizeof(float)>>>(
      inputSize, inputChannels,
      outputChannels, dInput, dOutput, 
      dFilter1, dFilter2, filterSize
   );
   // printf("toeplitz ended\n");

   status = hipMemcpy(hOutput.data(), dOutput, hOutput.size()*sizeof(float), hipMemcpyDeviceToHost);
   if (check_error_status(status, "couldn't load device output to host"))
      return false;
   printf("Output toeplitz:\n");
   printVector2d(hOutput, inputSize, 0);
   // printVector2d(hOutput, inputSize, 1);

   hipFree(dInput);
   hipFree(dOutput);
   hipFree(dFilter1);
   hipFree(dFilter2);
   return true;
}

void test1() {
   // const int inputChannels = 3;
	// const int outputChannelsArr[]{3, 8, 16, 32, 64, 128, 256};
	// const int filterSize = 3;
   // // mini_test();   
   // const int testNum = 10;
   // const int inputSizeMax = 1 << 10;
   // ofstream res("test_res.txt");   
   // res << "test num:" << testNum << endl;

   // for (int outputChannels : outputChannelsArr) {
   //    for (int inputSize = 256; inputSize <= inputSizeMax; inputSize <<= 1) {
   //       vector<float> hInput(inputChannels*(inputSize+2)*(inputSize+2));
   //       vector<float> hOutput(outputChannels*inputSize*inputSize);
   //       vector<float> hOutputToeplitz(outputChannels*inputSize*inputSize);
   //       vector<float> hFilter1(inputChannels*filterSize*filterSize);
   //       vector<float> hFilter2(inputChannels*outputChannels);

   //       res << "input:output(size*size) " << inputChannels << ":" << outputChannels << "(" 
   //          << inputSize << "*" << inputSize << ")" << endl;
   //       printf("input:output(size*size) %d:%d(%d*%d)\n\t", inputChannels, outputChannels, inputSize, inputSize);
         

   //       float funcTime = 0, memTime = 0;
   //       for (int i = 1; i <= testNum; i++) {
   //          fillInput(hInput, inputChannels, inputSize);
   //          fillFilter1(hFilter1, inputChannels, filterSize);
   //          fillFilter2(hFilter2, inputChannels, outputChannels);
   //          test(inputSize, inputChannels, outputChannels, filterSize,
   //             hInput, hOutput, hFilter1, hFilter2, 
   //             funcTime, memTime); 
   //          // test_toeplitz(inputSize, inputChannels, outputChannels, filterSize,
   //          //    hInput, hOutputToeplitz, hFilter1, hFilter2, 
   //          //    funcTime, memTime);
   //          printf("%d.pass  ", i);
   //          if (i% 10 == 0)
   //             printf("\n");
   //       }
   //       res << "   function: " << funcTime << "ms; " << "memory: " << memTime << "ms" << endl;
   //       res << "   function avg: " << funcTime / testNum << "ms; " << "memory avg: " << memTime / testNum << "ms" << endl; 
   //    }  
   //    res << endl;
   // }
   // res.close();
}

void test2() {
   const int inputChannels = 3;
   const int outputChannels = 16;
   const int inputSize = 256;
   const int filterSize = 3;
   const int testNum = 100;

   vector<float> hInput(inputChannels*(inputSize+2)*(inputSize+2));
   vector<float> hOutput(outputChannels*inputSize*inputSize);
   vector<float> hOutputToeplitz(outputChannels*inputSize*inputSize);
   vector<float> hFilter1(inputChannels*filterSize*filterSize);
   vector<float> hFilter2(inputChannels*outputChannels);


   for (int i = 1; i <= testNum; i++) {
      fillInput(hInput, inputChannels, inputSize);
      fillFilter1(hFilter1, inputChannels, filterSize);
      fillFilter2(hFilter2, inputChannels, outputChannels);
      test(inputSize, inputChannels, outputChannels, filterSize,
         hInput, hOutput, hFilter1, hFilter2
      );
      test_toeplitz(inputSize, inputChannels, outputChannels, filterSize,
         hInput, hOutputToeplitz, hFilter1, hFilter2
      );
      int size = hOutput.size();
      for (int idx = 0; idx < size; idx++) {
         if (hOutput[idx] != hOutputToeplitz[idx]) {
            printf("%d. fail\n", i);
            printf("Standart output: %f;  Toeplitz output:%f;\n", hOutput[idx], hOutputToeplitz[idx]);
            printf("idx=%d", idx);
         }
      }

      printf("%d. pass  ", i);
      if (i % 10 == 0) {
         printf("\n");
      }
   }
}

void unetx1() {
   // depthwise_conv2d_68 - conv2d_115
   // conv_1_1
   const int inputSize = 256;
   const int filterSize = 3;
   const int inputChannels = 3;
   const int outputChannels = 16;
   const int testNum = 1000;
   printf("UNET X: depthwise_conv2d_68 - const2d_115\n");
   printf("input:output(size*size) %d:%d(%d*%d)\n\t", inputChannels, outputChannels, inputSize, inputSize);

   vector<float> hInput(inputChannels*(inputSize+2)*(inputSize+2));
   vector<float> hOutput(outputChannels*inputSize*inputSize);
   vector<float> hOutputToeplitz(outputChannels*inputSize*inputSize);
   vector<float> hFilter1(inputChannels*filterSize*filterSize);
   vector<float> hFilter2(inputChannels*outputChannels);

   for (int i = 1; i <= testNum; i++) {
      fillInput(hInput, inputChannels, inputSize);
      fillFilter1(hFilter1, inputChannels, filterSize);
      fillFilter2(hFilter2, inputChannels, outputChannels);
      test(inputSize, inputChannels, outputChannels, filterSize,
         hInput, hOutput, hFilter1, hFilter2
      );
      test_toeplitz(inputSize, inputChannels, outputChannels, filterSize,
         hInput, hOutputToeplitz, hFilter1, hFilter2
      );
      int size = hOutput.size();
      for (int idx = 0; idx < size; idx++) {
         if (hOutput[idx] != hOutputToeplitz[idx]) {
            printf("%d. fail\n", i);
            printf("Standart output: %f;  Toeplitz output:%f;\n", hOutput[idx], hOutputToeplitz[idx]);
            printf("idx=%d", idx);
         }
      }
      // printf("%d. pass  ", i);
      // if (i % 10 == 0) {
      //    printf("\n");
      // }
      if (i % 100 == 0) {
         printf("%d.pass  ", i);
      }
   }
}

void unetx2() {
   // depthwise_conv2d_72 - conv2d_119
   // conv_3_1

   const int inputSize = 64;
   const int filterSize = 3;
   const int inputChannels = 3;
   const int outputChannels = 32;
   const int testNum = 1000;
   printf("UNET X: depthwise_conv2d_72 - conv2d_119\n");
   printf("input:output(size*size) %d:%d(%d*%d)\n\t", inputChannels, outputChannels, inputSize, inputSize);

   vector<float> hInput(inputChannels*(inputSize+2)*(inputSize+2));
   vector<float> hOutput(outputChannels*inputSize*inputSize);
   vector<float> hOutputToeplitz(outputChannels*inputSize*inputSize);
   vector<float> hFilter1(inputChannels*filterSize*filterSize);
   vector<float> hFilter2(inputChannels*outputChannels);

   for (int i = 1; i <= testNum; i++) {
      fillInput(hInput, inputChannels, inputSize);
      fillFilter1(hFilter1, inputChannels, filterSize);
      fillFilter2(hFilter2, inputChannels, outputChannels);
      test(inputSize, inputChannels, outputChannels, filterSize,
         hInput, hOutput, hFilter1, hFilter2
      );
      test_toeplitz(inputSize, inputChannels, outputChannels, filterSize,
         hInput, hOutputToeplitz, hFilter1, hFilter2
      );
      int size = hOutput.size();
      for (int idx = 0; idx < size; idx++) {
         if (hOutput[idx] != hOutputToeplitz[idx]) {
            printf("%d. fail\n", i);
            printf("Standart output: %f;  Toeplitz output:%f;\n", hOutput[idx], hOutputToeplitz[idx]);
            printf("idx=%d", idx);
         }
      }
      // printf("%d. pass  ", i);
      // if (i % 10 == 0) {
      //    printf("\n");
      // }
      if (i % 100 == 0) {
         printf("%d.pass  ", i);
      }
   }
}

void unetx3() {
   // depthwise_conv2d_77 - conv2d_124
   // conv_5_2


   const int inputSize = 16;
   const int filterSize = 3;
   const int inputChannels = 3;
   const int outputChannels = 256;
   const int testNum = 1000;
   printf("UNET X: depthwise_conv2d_77 - conv2d_124\n");
   printf("input:output(size*size) %d:%d(%d*%d)\n\t", inputChannels, outputChannels, inputSize, inputSize);

   vector<float> hInput(inputChannels*(inputSize+2)*(inputSize+2));
   vector<float> hOutput(outputChannels*inputSize*inputSize);
   vector<float> hOutputToeplitz(outputChannels*inputSize*inputSize);
   vector<float> hFilter1(inputChannels*filterSize*filterSize);
   vector<float> hFilter2(inputChannels*outputChannels);

   for (int i = 1; i <= testNum; i++) {
      fillInput(hInput, inputChannels, inputSize);
      fillFilter1(hFilter1, inputChannels, filterSize);
      fillFilter2(hFilter2, inputChannels, outputChannels);
      test(inputSize, inputChannels, outputChannels, filterSize,
         hInput, hOutput, hFilter1, hFilter2
      );
      test_toeplitz(inputSize, inputChannels, outputChannels, filterSize,
         hInput, hOutputToeplitz, hFilter1, hFilter2
      );
      int size = hOutput.size();
      for (int idx = 0; idx < size; idx++) {
         if (hOutput[idx] != hOutputToeplitz[idx]) {
            printf("%d. fail\n", i);
            printf("Standart output: %f;  Toeplitz output:%f;\n", hOutput[idx], hOutputToeplitz[idx]);
            printf("idx=%d", idx);
         }
      }
      // printf("%d. pass  ", i);
      // if (i % 10 == 0) {
      //    printf("\n");
      // }
      if (i % 100 == 0) {
         printf("%d.pass  ", i);
      }
   }
}

int main() {
	// test1();
   // test2();
   unetx1();
   // unetx2();
   // unetx3();
}